#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <iostream>
using namespace std;

#include "CycleTimer.h"

#define DEBUG

#ifdef DEBUG
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", 
        hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#else
#define cudaCheckError(ans) ans
#endif

const int threadsPerBlock = 256;

// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// gpu_bitonic_sort
__global__ void gpu_bitonic_sort(int* input, int j, int k)
{	
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	int id = i ^ j;
	if(id > i){
		if((i & k) == 0){
			if(input[i] > input[id]){
				int val = input[id];
				input[id] = input[i];
				input[i] = val;
			}
		}
		else{
			if(input[i] < input[id]){
				int val = input[id];
				input[id] = input[i];
				input[i] = val;
			}
		}
	}
	
	
}


// cudaSorting
double cudaSorting(int* input, int* end, int* result)
{
    int* device_input;
    int N = end - input;  
	int roundN = nextPow2(N);
	size_t size = N * sizeof(int);
	size_t size2 = roundN * sizeof(int);
	
	int* tmp = (int*)malloc(size2);
	memcpy(tmp, input, size);
	for(int i=N; i<roundN; i++){
		tmp[i] = 2147483647;
	}

	cudaCheckError(hipMalloc(&device_input, size2));
	cudaCheckError(hipMemcpy(device_input, tmp, size2, hipMemcpyHostToDevice));



    double startTime = CycleTimer::currentSeconds();
	const int blocksPerGrid = (roundN + threadsPerBlock - 1) / threadsPerBlock;
	
	// BITONIC BEGIN
	
	for(int k=2; k<=roundN; k<<=1){
		for(int j=k>>1; j>0; j>>=1){
			gpu_bitonic_sort<<<blocksPerGrid, threadsPerBlock>>>(device_input, j, k);
			cudaCheckError(hipDeviceSynchronize());
		}
	}
	
	// BITONIC END
	cudaCheckError(hipMemcpy(result, device_input, size, hipMemcpyDeviceToHost));
	
	

	free(tmp);
	
	hipFree(device_input);
	double endTime = CycleTimer::currentSeconds();
    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaSortingThrust --
double cudaSortingThrust(int* input, int* end, int* output) {

	
    int N = end - input;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(N);
    
	size_t size = N * sizeof(int);
	
    cudaCheckError(hipMemcpy(d_input.get(), input, size, hipMemcpyHostToDevice));

    double startTime = CycleTimer::currentSeconds();

	thrust::sort(d_input, d_input + N);

    cudaCheckError(hipDeviceSynchronize());
    double endTime = CycleTimer::currentSeconds();
   
    cudaCheckError(hipMemcpy(output, d_input.get(), size, hipMemcpyDeviceToHost));

    thrust::device_free(d_input);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
