#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <iostream>
using namespace std;

#include "CycleTimer.h"

#define DEBUG

#ifdef DEBUG
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", 
        hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#else
#define cudaCheckError(ans) ans
#endif

const int threadsPerBlock = 256;

// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__device__ void gpu_merge(int* input, int* output, int left, int width, int N)
{
	int mid = min(left + (width >> 1), N);
	int right = min(left + width, N);
	
	int i = left;
	int j = mid;
	for(int k=left;k<right;k++){
		if(i < mid && (j >= right || input[i] < input[j])){
			output[k] = input[i];
			i++;
		}
		else{
			output[k] = input[j];
			j++;
		}
	}	
}

// gpu_merge_sort
__global__ void gpu_merge_sort(int* input, int* output, int width, int N)
{	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i <= N / width){
		gpu_merge(input, output, i * width, width, N);
	}
}


// cudaSorting
double cudaSorting(int* input, int* end, int* result)
{
    int* device_output;
    int* device_input;
    int N = end - input;  
	size_t size = N * sizeof(int);
	
	cudaCheckError(hipMalloc(&device_input, size));
	cudaCheckError(hipMalloc(&device_output, size));
	cudaCheckError(hipMemcpy(device_input, input, size, hipMemcpyHostToDevice));
	
	
	
    double startTime = CycleTimer::currentSeconds();
	
	// MERGESORt BEGIN
	const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	int roundN = nextPow2(N);
	int* tmp;
	for(int w = 2; w <= roundN; w <<= 1){
		gpu_merge_sort<<<blocksPerGrid, threadsPerBlock>>>(device_input, device_output, w, N);
		tmp = device_input;
		device_input = device_output;
		device_output = tmp;
		
		// Wait for completion
		cudaCheckError(hipDeviceSynchronize());
	}
	cudaCheckError(hipMemcpy(result, device_input, size, hipMemcpyDeviceToHost));
	// MERGESORt END


    double endTime = CycleTimer::currentSeconds();

	hipFree(device_input);
	hipFree(device_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaSortingThrust --
double cudaSortingThrust(int* input, int* end, int* output) {

	
    int N = end - input;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(N);
    
	size_t size = N * sizeof(int);
	
    cudaCheckError(hipMemcpy(d_input.get(), input, size, hipMemcpyHostToDevice));

    double startTime = CycleTimer::currentSeconds();

	thrust::sort(d_input, d_input + N);

    cudaCheckError(hipDeviceSynchronize());
    double endTime = CycleTimer::currentSeconds();
   
    cudaCheckError(hipMemcpy(output, d_input.get(), size, hipMemcpyDeviceToHost));

    thrust::device_free(d_input);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
