#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <iostream>
#include <cstdint>

using namespace std;

#include "CycleTimer.h"

#define DEBUG

#ifdef DEBUG
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr, "CUDA Error: %s at %s:%d\n", 
        hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#else
#define cudaCheckError(ans) ans
#endif

const int threadsPerBlock = 256;

// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// gpu_bitonic_sort
__global__ void gpu_bitonic_sort(uint32_t* input, int j, int k)
{	
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	int id = i ^ j;
	if(id > i){
		if((i & k) == 0){
			if(input[i] > input[id]){
				int val = input[id];
				input[id] = input[i];
				input[i] = val;
			}
		}
		else{
			if(input[i] < input[id]){
				int val = input[id];
				input[id] = input[i];
				input[i] = val;
			}
		}
	}
	
	
}

// sum of a[i] * i
__global__ void gpu_sum(uint32_t* input, uint32_t* result, int n)
{	
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < n){
		atomicAdd(result, input[i] * i);
	}
}

inline uint32_t encrypt(uint32_t m, uint32_t key) {
    return (m*m + key)%key;
}

uint32_t cudaFunc(int n, int k){
	int N = nextPow2(n);
	size_t size = N * sizeof(uint32_t);
	
	uint32_t* input = (uint32_t*)malloc(size);
	uint32_t* device_input;
	for(int i=0; i<n; i++){
		input[i] = encrypt(i, k);
	}
	for(int i=n; i<N; i++){
		input[i] = UINT32_MAX;
	}
	
	
	cudaCheckError(hipMalloc(&device_input, size));
	cudaCheckError(hipMemcpy(device_input, input, size, hipMemcpyHostToDevice));
	const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	
	// BITONIC BEGIN
	
	for(int k=2; k<=N; k<<=1){
		for(int j=k>>1; j>0; j>>=1){
			gpu_bitonic_sort<<<blocksPerGrid, threadsPerBlock>>>(device_input, j, k);
			cudaCheckError(hipDeviceSynchronize());
		}
	}
	// BITONIC END
	
	uint32_t* device_result;
	cudaCheckError(hipMalloc(&device_result, sizeof(uint32_t)));
	gpu_sum<<<blocksPerGrid, threadsPerBlock>>>(device_input, device_result, n);
	cudaCheckError(hipDeviceSynchronize());
	
	uint32_t* sum = (uint32_t*)malloc(sizeof(uint32_t));
	cudaCheckError(hipMemcpy(sum, device_result, sizeof(uint32_t), hipMemcpyDeviceToHost));
	hipFree(device_input);
	hipFree(device_result);
    return *sum;

}

// cudaSortingThrust --
double cudaSortingThrust(int* input, int* end, int* output) {

	
    int N = end - input;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(N);
    
	size_t size = N * sizeof(int);
	
    cudaCheckError(hipMemcpy(d_input.get(), input, size, hipMemcpyHostToDevice));

    double startTime = CycleTimer::currentSeconds();

	thrust::sort(d_input, d_input + N);

    cudaCheckError(hipDeviceSynchronize());
    double endTime = CycleTimer::currentSeconds();
   
    cudaCheckError(hipMemcpy(output, d_input.get(), size, hipMemcpyDeviceToHost));

    thrust::device_free(d_input);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
